//Juan Pablo Alonso Escobar
#include "setup.h"

namespace bt{

void setParameters(thrust::host_vector<parameters>& par){
	for (int i=0;i<par.size();i++){
		par[i].fastMA=20;
		par[i].slowMA=60;
		par[i].orderSize=1000;
		par[i].initEq=100000;
	}
}

void extractRawData(char* filename,thrust::host_vector<bt::stockData>& data,bool header){
	std::ifstream input(filename);
	std::string line;
	char date[20];
	char d[DATA_ELEMENTS][20];
	//looks for comma placements and line length
	int c[DATA_ELEMENTS],len;
	//ignore first line if header exists
	if (header)getline(input,line);
	//tracks line number
	long lineCount=0;
	bt::stockData tempData;
	//loop through each line and assign to vector
	while (!input.eof()){
		getline(input,line);

		c[0]=line.find(',',0);
		//get date
		if (c[0]>0){
			line.copy(date,c[0]);
			date[c[0]]='\0';
			strcpy(tempData.date,date);
		}
		else{
			cout<<"Last line parsed"<<endl;
			break;
		}

		for (int i=0;i<DATA_ELEMENTS;i++){
			//find location of next comma
			c[i+1]=line.find(',',c[i]+1);
			//get value between commas if comma exists
			if (c[i+1]>0){
				line.copy(d[i],c[i+1]-c[i]-1,c[i]+1);
				d[i][c[i+1]-c[i]-1]='\0';

			}
			//if there are no more commas, get last value and exit loop
			else {
				line.copy(d[i],len-c[i],c[i]+1);
				tempData.d[i]=atof(d[i]);
				break;
			}
			//transform to float and copy data to vector
			tempData.d[i]=atof(d[i]);
		}
		data.push_back(tempData);
	}
}

//namespace bt
}



