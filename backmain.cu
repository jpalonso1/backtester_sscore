#include "hip/hip_runtime.h"
//Juan Pablo Alonso
//GPU Backtester 1.0

#include "setup.h"
#include "gpu.h"
#include <ctime>

using namespace std;

void runBacktest(thrust::device_vector<bt::stockData>& data,
		thrust::device_vector<bt::parameters>& par, thrust::device_vector<bt::result>& res,
		long vecSize,int etf){
	//create "dummy" vector sequence. Only used to track position
	thrust::device_vector<long> Y(vecSize);
    thrust::sequence(Y.begin(),Y.end());
    //wrap data in device pointer
    bt::stockData* dataPtr=thrust::raw_pointer_cast(&data[0]);
    //transform the vector using the specified function
    thrust::transform(par.begin(), par.end(), Y.begin(), res.begin(),
			individual_run(dataPtr,data.size(),etf));
}

void optimizeParameters(thrust::device_vector<bt::result>& res){
	thrust::sort(res.begin(),res.end(),return_max());
}

void printOptimal( thrust::host_vector<bt::result> resh,int etf){
    for (int i=0;i<1;i++){
		cout<<etf<<" - Sum PnL: "<<resh[i].PnL[DATA_ELEMENTS];
		cout<<" sharpe: "<<resh[i].sharpe[DATA_ELEMENTS];
		cout<<" Max Draw: "<<resh[i].maxDrawdown[DATA_ELEMENTS]<<endl;
		cout<<" SBE: "<<resh[i].pars.fPar[bt::SBE][etf];
		cout<<" SBC: "<<resh[i].pars.fPar[bt::SBC][etf];
		cout<<" SSE: "<<resh[i].pars.fPar[bt::SSE][etf];
		cout<<" SSC: "<<resh[i].pars.fPar[bt::SSC][etf];
		cout<<" WindowSize: "<<resh[i].pars.lPar[bt::windowSize][etf]<<endl;
    }
}

void copyResult(bt::result& optRes,bt::result& lastRes,int etf){
	optRes.pars.fPar[bt::SBE][etf]=lastRes.pars.fPar[bt::SBE][etf];
	optRes.pars.fPar[bt::SBC][etf]=lastRes.pars.fPar[bt::SBC][etf];
	optRes.pars.fPar[bt::SSE][etf]=lastRes.pars.fPar[bt::SSE][etf];
	optRes.pars.fPar[bt::SSC][etf]=lastRes.pars.fPar[bt::SSC][etf];
	optRes.pars.fPar[bt::windowSize][etf]=lastRes.pars.fPar[bt::windowSize][etf];
}

int main(){
	//get data
	thrust::host_vector<bt::stockData> datah;
	bt::extractRawData(dataFile,datah,true);
	thrust::device_vector<bt::stockData>datad(datah.size());
//	thrust::device_vector<bt::stockData> datad=datah;
	thrust::copy(datah.begin(), datah.end(), datad.begin());

    bt::result optRes;
    int etf;
    for (etf=0;etf<35;etf++){
		//create vector of parameters to be tested
		thrust::host_vector<bt::parameters> parh;
		long VEC_SIZE=setParameters(parh,etf);
		cout<<"Number of simulations: "<<VEC_SIZE<<endl;
		thrust::device_vector<bt::parameters> pard(VEC_SIZE);
		thrust::copy(parh.begin(), parh.end(), pard.begin());
		//    thrust::device_vector<bt::parameters> pard=parh;
		thrust::device_vector<bt::result> resd(VEC_SIZE);
		thrust::host_vector<bt::result> resh(VEC_SIZE);

		setParameters(parh,etf);
		//run the backtesting on gpu
		runBacktest(datad,pard,resd,VEC_SIZE,etf);

		//sort on gpu
		optimizeParameters(resd);
		thrust::copy(resd.begin(), resd.end(), resh.begin());

		//update optimalRes
		copyResult(optRes,resh[0],etf);

		//sample output
		printOptimal(resh,etf);
    }




    clock_t timeEnd=clock();

    cout<<"returned s Scores: "<<optRes.temp<<endl;
    cout<<"returned s Scores: "<<optRes.temp<<endl;
    cout<<"Total Runtime (see README.txt): "<<double(timeEnd)/double(CLOCKS_PER_SEC)<<" seconds"<<endl;

    return 0;
}
