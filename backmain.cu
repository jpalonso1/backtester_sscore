#include "hip/hip_runtime.h"
//Juan Pablo Alonso Escobar
//GPU Backtester 1.0

#include "setup.h"
#include "gpu.h"
#include "xlog.h"

using namespace std;

void runBacktest(thrust::device_vector<bt::stockData>& data,
		thrust::device_vector<bt::parameters>& par, thrust::device_vector<bt::result>& res,
		long vecSize){
	//create "dummy" vector sequence. Only used to track position
	thrust::device_vector<long> Y(vecSize);
    thrust::sequence(Y.begin(),Y.end());
    //wrap data in device pointer
    bt::stockData* dataPtr=thrust::raw_pointer_cast(&data[0]);
    //transform the vector using the specified function
    thrust::transform(par.begin(), par.end(), Y.begin(), res.begin(),
			individual_run(dataPtr,data.size()));
    //optimize results.
//    thrust::sort(Y.begin(), Y.end(),custom_sort(dataPtr,data.size()));
}

void optimizeParameters(thrust::device_vector<bt::result>& res){
	thrust::sort(res.begin(),res.end(),return_max());
}

int main(){
	cout<<"starting"<<endl;
	XLog logMain("Main process");
	thrust::host_vector<bt::stockData> datah;
	XLog logExtract("Extracting data");
	bt::extractRawData(dataFile,datah,true);
	logExtract.log("Lines: ",datah.size());
	logExtract.end();

	//create vector of parameters to be tested
	thrust::host_vector<bt::parameters> parh;
	long VEC_SIZE=setParameters(parh);
    cout<<"Vector Size: "<<VEC_SIZE<<endl;

    thrust::device_vector<bt::parameters> pard=parh;
    thrust::device_vector<bt::stockData> datad=datah;
//    thrust::device_vector<bt::execution> exec(VEC_SIZE);
    thrust::device_vector<bt::result> res(VEC_SIZE);

    XLog logBacktest("Run backtest");
    logBacktest.start();
    runBacktest(datad
    		,pard,res,VEC_SIZE);
    logBacktest.end();


    XLog logSort("Sorting");
    logSort.start();
    optimizeParameters(res);
    logSort.end();

    thrust::host_vector<bt::result> resh=res;

//    cout<<exech[0].trade[0].location[0]<<endl;
//    cout<<exech[0].trade[0].posSize[0]<<endl;
    cout<<"Parameters vec size: "<<VEC_SIZE<<endl;
    for (int i=0;i<10;i++){
		cout<<i<<"Sum PnL: "<<resh[i].PnL[DATA_ELEMENTS];
		cout<<" sharpe: "<<resh[i].sharpe[DATA_ELEMENTS];
		cout<<" avgdailyProf: "<<resh[i].avgDailyProfit[DATA_ELEMENTS];
		cout<<" Max Draw: "<<resh[i].maxDrawdown[DATA_ELEMENTS]<<endl;
    }

    for (int sym=0;sym<DATA_ELEMENTS;sym++){
		cout<<"Single PnL: "<<resh[0].PnL[sym]<<endl;
    }
    logMain.end();
	return 0;
}
