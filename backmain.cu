#include "hip/hip_runtime.h"
//Juan Pablo Alonso Escobar
//GPU Backtester 1.0

#include "setup.h"
#include "gpu.h"
#include "xlog.h"

using namespace std;


__host__ __device__
void aggregateResults(bt::execution& exec,bt::stockData* data,long dataSize){
	//find the PnL for each execution IF it is closing an exisiting position
	long thisPos=0;
	long partSize=0;
	for (int sym=0;sym<DATA_ELEMENTS;sym++)
		{
		//when last position was closed or reversed
		long lastClose=0;
		//direction (1 buy, 0 sell)
		int dir;
		//current net positon
		long netPos=exec.trade[sym].posSize[0];
		long thisPos,targetFill;
		float thisPrice;
		long partialFill=0;
		bool closed;

//		//loop through each execution and update results
		for (long i=1;i<=exec.numTrades[sym];i++){
//		for (long i=1;i<min(10,int(exec.numTrades[sym]));i++){
			//set position,price and direction
			thisPos=exec.trade[sym].posSize[i];
			thisPrice=data[exec.trade[sym].location[i]].d[sym];
			if (thisPos>0)dir=1;
			else dir=0;
			//check if closing position (reducing abs net)
			if (thisPos*netPos<0)closed=true;
			else closed=false;
			//update net position
			netPos+=thisPos;

			//Check for position closed, capture profit
			if (closed==true){
				//check if trade exceeds current position (reversal) and adjust
				if (thisPos*netPos>0)thisPos-=netPos;
				targetFill=thisPos;
				float priceSum=0;
				int dirCheck=0;
				for(long j=lastClose;j<exec.numTrades[sym];j++){
					//get checked trade direction
					if (exec.trade[sym].posSize[j]>0)dirCheck=1;
					else dirCheck=0;
					//check direction. if trades do not cancel each other, continue
					if (dirCheck==dir)continue;

					float posCheck=exec.trade[sym].posSize[j]+partialFill;


					float priceCheck=data[exec.trade[sym].location[j]].d[sym];
					//check for enough shares at current trade
					if (abs(posCheck)>abs(targetFill)){
						priceSum+=targetFill*priceCheck;
						partialFill+=targetFill;
						targetFill=0;
						lastClose=j;
					}
					//if check is smaller, take entire position and move on
					else{
						priceSum+=posCheck*priceCheck;
						partialFill=0;
						targetFill+=posCheck;
						lastClose=j+1;
					}
					//check if position has been filled
					if (targetFill==0)break;
				}
				//clear partial if crossed
				if (thisPos*netPos>0)partialFill=0;
				float avgPrice=abs(priceSum/thisPos);
				//update pnl
				exec.trade[sym].realPnL[i]=thisPos*(avgPrice-thisPrice);
			}
			//add a "closer"execution that assumes position is
			//forcefully closed at last date
			if (((i+1)==exec.numTrades[sym]) && (netPos!=0)){
				exec.trade[sym].posSize[exec.numTrades[sym]]=-netPos;
				exec.trade[sym].location[exec.numTrades[sym]]=dataSize-1;
				exec.numTrades[sym]++;
			}
		}
	}

	//TEMP: print
//	for (int sym=0;sym<1;sym++){
//		float execPnL=0;
//		for (long i=0;i<(exec.numTrades[sym]);i++){
//			float thisPos=exec.trade[sym].posSize[i];
//			float thisPrice=data[exec.trade[sym].location[i]].d[sym];
//			execPnL+=exec.trade[sym].realPnL[i];
//			testOut<<i<<",price,"<<thisPrice<<",pos,"<<thisPos<<",PnL,"<<
//					exec.trade[sym].realPnL[i]<<",loc,"<<
//					exec.trade[sym].location[i]<<endl;
//		}
//
//	}
}

struct individual_run
{
	//hold a copy of the pointer to data
	bt::stockData* data;
	long dataSize;
    individual_run(bt::stockData* _data,long _dataSize) :
    	data(_data),dataSize(_dataSize) {}

    __host__ __device__
    bt::execution operator()(const bt::parameters& par, const long& Y) const {
    	//to be run every iteration of the backtest
    	bt::execution execTemp;
//    	execTemp.posSize[0]=par.orderSize+2;
//    	execTemp.location[0]=123;
    	execTemp.numTrades[0]=0;
    	crossingMA(data,dataSize,0,100.0,par.fastMA,par.slowMA,execTemp);
    	aggregateResults(execTemp,data,dataSize);
    	getStats(execTemp,data,dataSize);
    	return execTemp;
	}
};

void runBacktest(thrust::device_vector<bt::stockData>& data,
		thrust::device_vector<bt::parameters>& par, thrust::device_vector<bt::execution>& exec){
	thrust::device_vector<long> Y(100);
    thrust::sequence(Y.begin(),Y.end());
    //wrap data in device pointer
    bt::stockData* dataPtr=thrust::raw_pointer_cast(&data[0]);
    //transform the vector using the specified function
//    cout<<"running transform"<<endl;
    thrust::transform(par.begin(), par.end(), Y.begin(), exec.begin(),
			individual_run(dataPtr,data.size()));
}

int main(){
	cout<<"starting"<<endl;
	XLog logMain("Main process");
	thrust::host_vector<bt::stockData> datah;
	XLog logExtract("Extracting data");
	bt::extractRawData("AAPLclean.csv",datah,true);
	logExtract.log("Lines: ",datah.size());
	logExtract.end();
	const long VEC_SIZE=10000;
//	cout<<"Sample: "<<datah[0].date<<" + "<<datah[0].d1
//			<<" + "<<datah[0].d2<<" + "<<datah[0].d3<<endl;
	cout<<"Sample: "<<datah[0].	date<<" + "<<datah[0].d[0]
			<<" + "<<datah[0].d[1]<<" + "<<datah[0].d[2]<<endl;
	//create vector of parameters to be tested
	thrust::host_vector<bt::parameters> parh(VEC_SIZE);
	setParameters(parh);
    parh[0].fastMA=27;
    thrust::device_vector<bt::parameters> pard=parh;
    thrust::device_vector<bt::stockData> datad=datah;
    cout<<"data test: "<<datah[5].d[0]<<","<<datah[4].d[0]<<endl;
    thrust::device_vector<bt::execution> exec(VEC_SIZE);

    XLog logBacktest("Run backtest");
    runBacktest(datad,pard,exec);
    logBacktest.end();

    thrust::host_vector<bt::execution> exech=exec;

    cout<<exech[0].trade[0].location[0]<<endl;
    cout<<exech[0].trade[0].posSize[0]<<endl;
    cout<<exech[4].trade[0].location[0]<<endl;
    cout<<exech[4].trade[0].posSize[0]<<endl;
    logMain.end();
	return 0;
}

void testDemo(){

}
