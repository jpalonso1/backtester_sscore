#include "hip/hip_runtime.h"
//Juan Pablo Alonso Escobar
//GPU Backtester 1.0

#include "setup.h"
#include "gpu.h"
#include "xlog.h"
#include "custom.h"

using namespace std;

void runBacktest(thrust::device_vector<bt::stockData>& data,
		thrust::device_vector<bt::parameters>& par, thrust::device_vector<bt::execution>& exec,
		long vecSize){
	//create "dummy" vector sequence. Only used to track position
	thrust::device_vector<long> Y(vecSize);
    thrust::sequence(Y.begin(),Y.end());
    //wrap data in device pointer
    bt::stockData* dataPtr=thrust::raw_pointer_cast(&data[0]);
    //transform the vector using the specified function
    thrust::transform(par.begin(), par.end(), Y.begin(), exec.begin(),
			individual_run(dataPtr,data.size()));

}

int main(){
	cout<<"starting"<<endl;
	XLog logMain("Main process");
	thrust::host_vector<bt::stockData> datah;
	XLog logExtract("Extracting data");
	bt::extractRawData(dataFile,datah,true);
	logExtract.log("Lines: ",datah.size());
	logExtract.end();

	//create vector of parameters to be tested
	thrust::host_vector<bt::parameters> parh;
	long VEC_SIZE=setParameters(parh);;
    cout<<"Vector Size: "<<VEC_SIZE<<endl;

    thrust::device_vector<bt::parameters> pard=parh;
    thrust::device_vector<bt::stockData> datad=datah;
    thrust::device_vector<bt::execution> exec(VEC_SIZE);

    XLog logBacktest("Run backtest");
    logBacktest.start();
    runBacktest(datad,pard,exec,VEC_SIZE);
    logBacktest.end();

    thrust::host_vector<bt::execution> exech=exec;

    cout<<exech[0].trade[0].location[0]<<endl;
    cout<<exech[0].trade[0].posSize[0]<<endl;
    cout<<"Parameters vec size: "<<VEC_SIZE<<endl;
    cout<<"Sum PnL: "<<exech[0].resTotal.PnL<<endl;
    cout<<"Max Draw: "<<exech[0].resTotal.maxDrawdown<<endl;
    logMain.end();
	return 0;
}
