#include "hip/hip_runtime.h"
//Juan Pablo Alonso
//GPU Backtester 1.0

#include "setup.h"
#include "gpu.h"
#include <ctime>

using namespace std;

void runBacktest(thrust::device_vector<bt::stockData>& data,
		thrust::device_vector<bt::parameters>& par, thrust::device_vector<bt::result>& res,
		long vecSize,int etf,bool totalRun=false){
	//create "dummy" vector sequence. Only used to track position
	thrust::device_vector<long> Y(vecSize);
    thrust::sequence(Y.begin(),Y.end());
    //wrap data in device pointer
    bt::stockData* dataPtr=thrust::raw_pointer_cast(&data[0]);
    //transform the vector using the specified function
    thrust::transform(par.begin(), par.end(), Y.begin(), res.begin(),
			individual_run(dataPtr,data.size(),etf));
}

void optimizeParameters(thrust::device_vector<bt::result>& res){
	thrust::sort(res.begin(),res.end(),retdraw_max());
}

void printOptimal(bt::result resh,int etf){
	cout<<etf<<" - Sum PnL: "<<resh.PnL[DATA_ELEMENTS];
	cout<<" sharpe: "<<resh.sharpe[DATA_ELEMENTS];
	cout<<" Max Draw: "<<resh.maxDrawdown[DATA_ELEMENTS]<<endl;
	cout<<" SBE: "<<resh.pars.fPar[bt::SBE][etf];
	cout<<" SBC: "<<resh.pars.fPar[bt::SBC][etf];
	cout<<" SSE: "<<resh.pars.fPar[bt::SSE][etf];
	cout<<" SSC: "<<resh.pars.fPar[bt::SSC][etf];
	cout<<" WindowSize: "<<resh.pars.lPar[bt::windowSize][etf]<<endl;
}

void printParameter(bt::parameters pars){
	parametersOut<<"sym,orderSize,SBE,SBC,SSE,SSC,windowSize"<<endl;
	for (int etf=0;etf<35;etf++){
		parametersOut<<etf<<","<<pars.lPar[bt::orderSize][etf]
		<<","<<pars.fPar[bt::SBE][etf]
		<<","<<pars.fPar[bt::SBC][etf]
		<<","<<pars.fPar[bt::SSE][etf]
		<<","<<pars.fPar[bt::SSC][etf]
		<<","<<pars.lPar[bt::windowSize][etf]<<endl;

	}
}

void copyResult(bt::result& optRes,bt::result& lastRes,int etf){
	optRes.pars.lPar[bt::orderSize][etf]=lastRes.pars.lPar[bt::orderSize][etf];
	optRes.pars.fPar[bt::SBE][etf]=lastRes.pars.fPar[bt::SBE][etf];
	optRes.pars.fPar[bt::SBC][etf]=lastRes.pars.fPar[bt::SBC][etf];
	optRes.pars.fPar[bt::SSE][etf]=lastRes.pars.fPar[bt::SSE][etf];
	optRes.pars.fPar[bt::SSC][etf]=lastRes.pars.fPar[bt::SSC][etf];
	optRes.pars.lPar[bt::windowSize][etf]=lastRes.pars.lPar[bt::windowSize][etf];
}

int main(){
	//get data
	thrust::host_vector<bt::stockData> datah;
	bt::extractRawData(iSample,datah,true);
	thrust::device_vector<bt::stockData>datad(datah.size());
//	thrust::device_vector<bt::	stockData> datad=datah;
	thrust::copy(datah.begin(), datah.end(), datad.begin());

	long VEC_SIZE;
    bt::result optRes;
    int etf;
    for (etf=0;etf<35;etf++){
		//create vector of parameters to be tested
		thrust::host_vector<bt::parameters> parh;
		VEC_SIZE=setParameters(parh,etf);
		cout<<"Number of simulations: "<<VEC_SIZE<<endl;
		thrust::device_vector<bt::parameters> pard(VEC_SIZE);
		thrust::copy(parh.begin(), parh.end(), pard.begin());
		//    thrust::device_vector<bt::parameters> pard=parh;
		thrust::device_vector<bt::result> resd(VEC_SIZE);
		thrust::host_vector<bt::result> resh(VEC_SIZE);

		//run the backtesting on gpu
		runBacktest(datad,pard,resd,VEC_SIZE,etf);

		//sort on gpu
		optimizeParameters(resd);
		thrust::copy(resd.begin(), resd.end(), resh.begin());

		//update optimalRes
		copyResult(optRes,resh[0],etf);

		//sample output
		printOptimal(resh[0],etf);
    }


    for (etf=0;etf<1;etf++){
		//create vector of parameters to be tested
		thrust::host_vector<bt::parameters> parh;
		cout<<"Number of simulations: "<<VEC_SIZE<<endl;
		thrust::device_vector<bt::parameters> pard(1);

		parh.push_back(optRes.pars);

		thrust::copy(parh.begin(), parh.end(), pard.begin());
		//    thrust::device_vector<bt::parameters> pard=parh;
		thrust::device_vector<bt::result> resd(1);
		thrust::host_vector<bt::result> resh(1);

		//run the backtesting on gpu
		runBacktest(datad,pard,resd,1,-1);

		//sort on gpu
		optimizeParameters(resd);
		thrust::copy(resd.begin(), resd.end(), resh.begin());

		//update optimalRes
		copyResult(optRes,resh[0],etf);

		//sample output
		printOptimal(resh[0],etf);
    }

    printParameter(optRes.pars);

    clock_t timeEnd=clock();

    cout<<"returned s Scores: "<<optRes.temp<<endl;
    cout<<"returned s Scores: "<<optRes.temp<<endl;
    cout<<"Total Runtime: "<<double(timeEnd)/double(CLOCKS_PER_SEC)<<" seconds"<<endl;

    return 0;
}
