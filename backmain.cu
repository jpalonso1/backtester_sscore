#include "hip/hip_runtime.h"
//Juan Pablo Alonso
//GPU Backtester 1.0

#include "setup.h"
#include "gpu.h"
#include "xlog.h"

using namespace std;

void runBacktest(thrust::device_vector<bt::stockData>& data,
		thrust::device_vector<bt::parameters>& par, thrust::device_vector<bt::result>& res,
		long vecSize){
	//create "dummy" vector sequence. Only used to track position
	thrust::device_vector<long> Y(vecSize);
    thrust::sequence(Y.begin(),Y.end());
    //wrap data in device pointer
    bt::stockData* dataPtr=thrust::raw_pointer_cast(&data[0]);
    //transform the vector using the specified function
    thrust::transform(par.begin(), par.end(), Y.begin(), res.begin(),
			individual_run(dataPtr,data.size()));

}

void optimizeParameters(thrust::device_vector<bt::result>& res){
	thrust::sort(res.begin(),res.end(),sharpe_max());
}

int main(){
	XLog logMain("Total time");

	//get data
	XLog logExtract("Extracting data");
	logExtract.start();
	thrust::host_vector<bt::stockData> datah;
	bt::extractRawData(dataFile,datah,true);
	thrust::device_vector<bt::stockData>datad(datah.size());
//	thrust::device_vector<bt::stockData> datad=datah;
	thrust::copy(datah.begin(), datah.end(), datad.begin());
	logExtract.log("Lines: ",datah.size());
	logExtract.end();

	//create vector of parameters to be tested
	XLog logPar("Setting parameters");
	logPar.start();
	thrust::host_vector<bt::parameters> parh;
	long VEC_SIZE=setParameters(parh);
	thrust::device_vector<bt::parameters> pard(VEC_SIZE);
	thrust::copy(parh.begin(), parh.end(), pard.begin());
	//    thrust::device_vector<bt::parameters> pard=parh;
    thrust::device_vector<bt::result> resd(VEC_SIZE);


    //run the backtesting on gpu
    XLog logBacktest("Run backtest");
    logBacktest.start();
    logBacktest.log("Total simulations to run: ",VEC_SIZE);
    runBacktest(datad
    		,pard,resd,VEC_SIZE);
    logBacktest.end();

    //sort on gpu
    XLog logSort("Sorting");
    logSort.start();
    optimizeParameters(resd);
    thrust::host_vector<bt::result> resh(resd.size());
    thrust::copy(resd.begin(), resd.end(), resh.begin());
    logSort.end();

    //sample output
    for (int i=0;i<10;i++){
		cout<<i<<" - Sum PnL: "<<resh[i].PnL[DATA_ELEMENTS];
		cout<<" sharpe: "<<resh[i].sharpe[DATA_ELEMENTS];
		cout<<" Max Drawdown: "<<resh[i].maxDrawdown[DATA_ELEMENTS]<<endl;
    }
    logMain.end();
	return 0;
}
