//Juan Pablo Alonso
//GPU Backtester 1.0

#include "setup.h"
#include "gpu.h"

using namespace std;

void runBacktest(thrust::device_vector<bt::stockData>& data,
		thrust::device_vector<bt::parameters>& par, thrust::device_vector<bt::result>& res,
		long vecSize){
	//create "dummy" vector sequence. Only used to track position
	thrust::device_vector<long> Y(vecSize);
    thrust::sequence(Y.begin(),Y.end());
    //wrap data in device pointer
    bt::stockData* dataPtr=thrust::raw_pointer_cast(&data[0]);
    //transform the vector using the specified function
    thrust::transform(par.begin(), par.end(), Y.begin(), res.begin(),
			individual_run(dataPtr,data.size()));
}

void optimizeParameters(thrust::device_vector<bt::result>& res){
	thrust::sort(res.begin(),res.end(),sharpe_max());
}

void printOptimal( thrust::host_vector<bt::result> resh){
    for (int i=0;i<10;i++){
		cout<<i<<" - Sum PnL: "<<resh[i].PnL[DATA_ELEMENTS];
		cout<<" sharpe: "<<resh[i].sharpe[DATA_ELEMENTS];
		cout<<" Max Draw: "<<resh[i].maxDrawdown[DATA_ELEMENTS]<<endl;
		cout<<" SBE: "<<resh[i].pars.fPar[bt::SBE];
		cout<<" SBC: "<<resh[i].pars.fPar[bt::SBC];
		cout<<" SSE: "<<resh[i].pars.fPar[bt::SSE];
		cout<<" SSC: "<<resh[i].pars.fPar[bt::SSC];
		cout<<" WindowSize: "<<resh[i].pars.lPar[bt::windowSize]<<endl;
    }
}

int main(){
	//get data
	thrust::host_vector<bt::stockData> datah;
	bt::extractRawData(dataFile,datah,true);
	thrust::device_vector<bt::stockData>datad(datah.size());
//	thrust::device_vector<bt::stockData> datad=datah;
	thrust::copy(datah.begin(), datah.end(), datad.begin());

	//create vector of parameters to be tested
	thrust::host_vector<bt::parameters> parh;
	long VEC_SIZE=setParameters(parh);
	cout<<"Number of simulations: "<<VEC_SIZE<<endl;
	thrust::device_vector<bt::parameters> pard(VEC_SIZE);
	thrust::copy(parh.begin(), parh.end(), pard.begin());
	//    thrust::device_vector<bt::parameters> pard=parh;
    thrust::device_vector<bt::result> resd(VEC_SIZE);


    //run the backtesting on gpu
    runBacktest(datad
    		,pard,resd,VEC_SIZE);

    //sort on gpu
    optimizeParameters(resd);
    thrust::host_vector<bt::result> resh(resd.size());
    thrust::copy(resd.begin(), resd.end(), resh.begin());

    //sample output
    printOptimal(resh);

    return 0;
}
