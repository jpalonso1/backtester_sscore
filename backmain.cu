#include "hip/hip_runtime.h"
//Juan Pablo Alonso Escobar
//GPU Backtester 1.0

#include "setup.h"
#include "gpu.h"
#include "xlog.h"
#include "custom.h"

using namespace std;

void runBacktest(thrust::device_vector<bt::stockData>& data,
		thrust::device_vector<bt::parameters>& par, thrust::device_vector<bt::result>& res,
		long vecSize){
	//create "dummy" vector sequence. Only used to track position
	thrust::device_vector<long> Y(vecSize);
    thrust::sequence(Y.begin(),Y.end());
    //wrap data in device pointer
    bt::stockData* dataPtr=thrust::raw_pointer_cast(&data[0]);
    //transform the vector using the specified function
    thrust::transform(par.begin(), par.end(), Y.begin(), res.begin(),
			individual_run(dataPtr,data.size()));
    //optimize results.
//    thrust::sort(Y.begin(), Y.end(),custom_sort(dataPtr,data.size()));
}

int main(){
	cout<<"starting"<<endl;
	XLog logMain("Main process");
	thrust::host_vector<bt::stockData> datah;
	XLog logExtract("Extracting data");
	bt::extractRawData(dataFile,datah,true);
	logExtract.log("Lines: ",datah.size());
	logExtract.end();

	//create vector of parameters to be tested
	thrust::host_vector<bt::parameters> parh;
	long VEC_SIZE=setParameters(parh);;
    cout<<"Vector Size: "<<VEC_SIZE<<endl;

    thrust::device_vector<bt::parameters> pard=parh;
    thrust::device_vector<bt::stockData> datad=datah;
//    thrust::device_vector<bt::execution> exec(VEC_SIZE);
    thrust::device_vector<bt::result> res(VEC_SIZE);

    XLog logBacktest("Run backtest");
    logBacktest.start();
    runBacktest(datad,pard,res,VEC_SIZE);
    logBacktest.end();

    thrust::host_vector<bt::result> resh=res;

//    cout<<exech[0].trade[0].location[0]<<endl;
//    cout<<exech[0].trade[0].posSize[0]<<endl;
    cout<<"Parameters vec size: "<<VEC_SIZE<<endl;
    cout<<"Sum PnL: "<<resh[0].PnL[0]<<endl;
    cout<<"Max Draw: "<<resh[0].maxDrawdown[0]<<endl;
    logMain.end();
	return 0;
}
